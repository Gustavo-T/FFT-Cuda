#include "hip/hip_runtime.h"
//**********************************
//Nathan Durst
//FFT Cuda Program
//December, 5 2016
//**********************************
//This application uses cuda c and implements
// the Cooley-Tukey FFT algorithm to transforms 
// an array of complex numbers into a data set
// correlation of complex numbers.
#include <stdio.h>
#include <math.h>
#define N 16384
#define PI 3.14

//kernel function declaration
__global__ void FFT(float * R, float * I, float * xR, float * xI);

int main()
{
	float R[N] = {0};
	float I[N] = {0};
	float xR[N], xI[N], *Rd, *Id, *xRd, *xId, elapsed;
	int i, size = N * sizeof(int);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(stop);
	hipEventRecord(start, 0);
	
	//initialize arrays of real and imaginary numbers
	R[0] = 3.6; R[1] = 2.9; R[2] = 5.6; R[3] = 4.8;
	R[4] = 3.3; R[5] = 5.9; R[6] = 5.0; R[7] = 4.3;
	I[0] = 2.6; I[1] = 6.3; I[2] = 4.0; I[3] = 9.1;
	I[4] = 0.4; I[5] = 4.8; I[6] = 2.6; I[7] = 4.1;
	
	//allocate size of arrays on device and store them in 
	// specified array variable names
	hipMalloc((void**)&Rd, size);
	hipMalloc((void**)&Id, size);
	hipMalloc((void**)&xRd, size);
	hipMalloc((void**)&xId, size);
	
	//copy initialized arrays to arrays on device
	hipMemcpy(Rd, R, size, hipMemcpyHostToDevice);
	hipMemcpy(Id, I, size, hipMemcpyHostToDevice);
	
	//determine dimensions of block and threads used by the kernel
	dim3 dimGrid((N/1024),1);
	dim3 dimBlock(1024, 1);

	//call kernel function FFT
	FFT<<<dimGrid, dimBlock>>>(Rd, Id, xRd, xId);
	
	//copy results from device arrays to the host arrays
	hipMemcpy(xR, xRd, size, hipMemcpyDeviceToHost);
	hipMemcpy(xI, xId, size, hipMemcpyDeviceToHost);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	//print results
	for (i = 0; i < 8; i++)
		printf("X[%d]: %0.1f + %0.1fi\n", i, xR[i], xI[i]);
	printf("The elapsed time of the program was %.2f ms\n", elapsed);
		
	//free space on device
	hipFree(Rd);
	hipFree(Id);
	hipFree(xRd);
	hipFree(xId);
}

__global__ void FFT(float * R, float * I, float * xR, float * xI)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	float real = 0, imag = 0;
	
	//iterate through entire array for each index and calculate even
	// and odd for real and imaginary numbers.
	for (int i = 0; i<(N/2); i++)
	{
		//even
		real += R[i] * cos((2*PI*(i*2))/N) - I[i] * sin((2*PI*id*(i*2))/N);
		imag += R[i] * -sin((2*PI*(i*2))/N) + I[i] * cos((2*PI*id*(i*2))/N);
		
		//odd
		real += R[i] * cos((2*PI*(i*2+1))/N) - I[i] * sin((2*PI*id*(i*2+1))/N);
		imag += R[i] * -sin((2*PI*(i*2+1))/N) + I[i] * cos((2*PI*id*(i*2+1))/N);
	}
	xR[id] = real;
	xI[id] = imag;
}